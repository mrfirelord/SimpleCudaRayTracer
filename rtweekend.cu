#include "hip/hip_runtime.h"
#ifndef RTWEEKEND_H
#define RTWEEKEND_H

#include <iostream>
#include <limits>
#include <memory>
#include <hiprand/hiprand_kernel.h>


// CUDA Function Decorators
#define C_DH __device__ __host__
#define C_D __device__
#define C_H __host__

// C++ Std Usings

using std::make_shared;
using std::shared_ptr;

// Constants

#define CHECK_CUDA(call) \
do { \
hipError_t err = call; \
if (err != hipSuccess) { \
std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(err) << std::endl; \
exit(1); \
} \
} while(0)

namespace rt_in_one_weekend {
    __device__ constexpr double infinity = std::numeric_limits<double>::infinity();
    __device__ constexpr double pi = 3.1415926535897932385;

    // Utility Functions

    C_DH inline double degreesToRadians(const double degrees) { return degrees * pi / 180.0; }

    C_D inline double randomDouble(hiprandState *state) {
        return hiprand_uniform_double(state);
    }

    C_D inline double randomDouble(hiprandState *state, double min, double max) {
        return min + (max - min) * hiprand_uniform_double(state);
    }
}

// Common Headers

#include "Vec3.cu"
#include "interval.cu"
#include "ray.cu"

#endif
