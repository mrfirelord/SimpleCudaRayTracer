#include "hip/hip_runtime.h"
#ifndef RTWEEKEND_H
#define RTWEEKEND_H

#include <iostream>
#include <limits>
#include <memory>
#include <hiprand/hiprand_kernel.h>


// C++ Std Usings

using std::make_shared;
using std::shared_ptr;

// Constants

#define CHECK_CUDA(call) \
do { \
hipError_t err = call; \
if (err != hipSuccess) { \
std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(err) << std::endl; \
exit(1); \
} \
} while(0)

namespace rt_in_one_weekend {
    __device__ constexpr double infinity = std::numeric_limits<double>::infinity();
    __device__ constexpr double pi = 3.1415926535897932385;

    // Utility Functions

    __device__ __host__ inline double degreesToRadians(const double degrees) { return degrees * pi / 180.0; }

    __device__ inline double randomDouble(hiprandState *state) {
        return hiprand_uniform_double(state);
    }

    __device__ inline double randomDouble(hiprandState *state, double min, double max) {
        return min + (max - min) * hiprand_uniform_double(state);
    }
}

// Common Headers

#include "Vec3.cu"
#include "interval.cu"
#include "ray.cu"

#endif
