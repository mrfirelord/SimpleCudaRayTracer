#include "hip/hip_runtime.h"
#ifndef VEC3_H
#define VEC3_H

#include <cmath>
#include <iostream>
#include <hip/hip_runtime.h>

namespace rt_in_one_weekend {
    class Vec3 {
    public:
        double e[3];

        __device__ __host__ Vec3() : e{0, 0, 0} {
        }

        __device__ __host__ Vec3(double e0, double e1, double e2) : e{e0, e1, e2} {
        }

        __device__ __host__ double x() const { return e[0]; }
        __device__ __host__ double y() const { return e[1]; }
        __device__ __host__ double z() const { return e[2]; }

        __device__ __host__ Vec3 operator-() const { return Vec3(-e[0], -e[1], -e[2]); }
        __device__ __host__ double operator[](int i) const { return e[i]; }
        __device__ __host__ double &operator[](int i) { return e[i]; }

        __device__ __host__ Vec3 &operator+=(const Vec3 &v) {
            e[0] += v.e[0];
            e[1] += v.e[1];
            e[2] += v.e[2];
            return *this;
        }

        __device__ __host__ Vec3 &operator*=(double t) {
            e[0] *= t;
            e[1] *= t;
            e[2] *= t;
            return *this;
        }

        __device__ __host__ Vec3 &operator/=(double t) {
            return *this *= 1 / t;
        }

        __device__ __host__ double length() const {
            return sqrt(lengthSquared());
        }

        __device__ __host__ double lengthSquared() const {
            return e[0] * e[0] + e[1] * e[1] + e[2] * e[2];
        }

        __device__ static Vec3 random(hiprandState *state) {
            return Vec3(randomDouble(state), randomDouble(state), randomDouble(state));
        }

        __device__ static Vec3 random(hiprandState *state, const double min, const double max) {
            return Vec3(randomDouble(state, min, max), randomDouble(state, min, max), randomDouble(state, min, max));
        }
    };

    // point3 is just an alias for Vec3, but useful for geometric clarity in the code.
    using Point3 = Vec3;

    // Vector Utility Functions

    __device__ __host__ inline std::ostream &operator<<(std::ostream &out, const Vec3 &v) {
        return out << v.e[0] << ' ' << v.e[1] << ' ' << v.e[2];
    }

    __device__ __host__ inline Vec3 operator+(const Vec3 &u, const Vec3 &v) {
        return Vec3(u.e[0] + v.e[0], u.e[1] + v.e[1], u.e[2] + v.e[2]);
    }

    __device__ __host__ inline Vec3 operator-(const Vec3 &u, const Vec3 &v) {
        return Vec3(u.e[0] - v.e[0], u.e[1] - v.e[1], u.e[2] - v.e[2]);
    }

    __device__ __host__ inline Vec3 operator*(const Vec3 &u, const Vec3 &v) {
        return Vec3(u.e[0] * v.e[0], u.e[1] * v.e[1], u.e[2] * v.e[2]);
    }

    __device__ __host__ inline Vec3 operator*(double t, const Vec3 &v) {
        return Vec3(t * v.e[0], t * v.e[1], t * v.e[2]);
    }

    __device__ __host__ inline Vec3 operator*(const Vec3 &v, double t) {
        return t * v;
    }

    __device__ __host__ inline Vec3 operator/(const Vec3 &v, double t) {
        return (1 / t) * v;
    }

    __device__ __host__ inline double dot(const Vec3 &u, const Vec3 &v) {
        return u.e[0] * v.e[0]
               + u.e[1] * v.e[1]
               + u.e[2] * v.e[2];
    }

    __device__ __host__ inline Vec3 cross(const Vec3 &u, const Vec3 &v) {
        return Vec3(u.e[1] * v.e[2] - u.e[2] * v.e[1],
                    u.e[2] * v.e[0] - u.e[0] * v.e[2],
                    u.e[0] * v.e[1] - u.e[1] * v.e[0]);
    }

    __device__ __host__ inline Vec3 unitVector(const Vec3 &v) {
        return v / v.length();
    }

    __device__ inline Vec3 randomUnitVector(hiprandState *state) {
        while (true) {
            auto p = Vec3::random(state, -1, 1);
            auto lengthSquared = p.lengthSquared();
            if (1e-160 < lengthSquared && lengthSquared <= 1)
                return p / sqrt(lengthSquared);
        }
    }

    __device__ inline Vec3 randomOnHemisphere(const Vec3 &normal, hiprandState *state) {
        Vec3 onUnitSphere = randomUnitVector(state);
        if (dot(onUnitSphere, normal) > 0.0) // In the same hemisphere as the normal
            return onUnitSphere;
        return -onUnitSphere;
    }
}
#endif
