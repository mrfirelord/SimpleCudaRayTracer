#include "hip/hip_runtime.h"
#ifndef CAMERA_H
#define CAMERA_H

#include "Vec3.cu"
#include "interval.cu"
#include "material.cu"

namespace rt_in_one_weekend {
    class Camera {
    public:
        int imageWidth;
        int imageHeight;
        int imageSize;
        double aspectRatio;
        double focalLength = 1.0;
        Point3 center;
        Vec3 pixel00Loc;
        Vec3 pixelDeltaU;
        Vec3 pixelDeltaV;
        int samplesPerPixel = 50; // Count of random samples for each pixel
        double pixelSamplesScale;
        unsigned int maxDepth = 50; // Maximum number of ray bounces into scene
        double vfov; // Vertical view angle (field of view)

        Point3 lookfrom; // Point camera is looking from
        Point3 lookat; // Point camera is looking at
        Vec3 vup = Vec3(0, 1, 0); // Camera-relative "up" direction

        Vec3 u, v, w; // Camera frame basis vectors

    public:
        Camera(const int imageWidth,
               const double aspectRatio,
               const unsigned int samplesPerPixel,
               const double vfov,
               Point3 lookfrom,
               Point3 lookAt): imageWidth(imageWidth),
                               aspectRatio(aspectRatio),
                               samplesPerPixel(samplesPerPixel),
                               vfov(vfov),
                               lookfrom(lookfrom),
                               lookat(lookAt) {
            imageHeight = static_cast<int>(imageWidth / aspectRatio);
            imageSize = imageWidth * imageHeight;

            center = lookfrom;
            focalLength = (lookfrom - lookat).length();

            auto theta = degreesToRadians(vfov);
            auto h = std::tan(theta / 2);
            const auto viewportHeight = 2 * h * focalLength;
            const auto viewportWidth = viewportHeight * (static_cast<double>(imageWidth) / imageHeight);

            // Calculate the u,v,w unit basis vectors for the camera coordinate frame.
            w = unitVector(lookfrom - lookat);
            u = unitVector(cross(vup, w));
            v = cross(w, u);

            // Calculate the vectors across the horizontal and down the vertical viewport edges.
            const auto viewportU = viewportWidth * u;
            const auto viewportV = viewportHeight * v;

            // Calculate the horizontal and vertical delta vectors from pixel to pixel.
            pixelDeltaU = viewportU / imageWidth;
            pixelDeltaV = viewportV / imageHeight;

            // Calculate the location of the upper left pixel.
            auto viewportUpperLeft = center - (focalLength * w) - viewportU / 2 - viewportV / 2;
            pixel00Loc = viewportUpperLeft + 0.5 * (pixelDeltaU + pixelDeltaV);
            pixelSamplesScale = 1.0 / samplesPerPixel;
        }
    };

    __device__ Vec3 sampleSquare(hiprandState *state) {
        // Returns the vector to a random point in the [-.5,-.5]-[+.5,+.5] unit square.
        return Vec3(randomDouble(state) - 0.5, randomDouble(state) - 0.5, 0);
    }

    __device__ Ray getRay(const int x, const int y, const Camera &camera, hiprandState *state) {
        // Construct a camera ray originating from the origin and directed at randomly sampled
        // point around the pixel location i, j.

        const auto offset = sampleSquare(state);
        const auto pixelSample = camera.pixel00Loc
                                 + (x + offset.x()) * camera.pixelDeltaU
                                 + (y + offset.y()) * camera.pixelDeltaV;

        const auto rayOrigin = camera.center;
        const auto rayDirection = pixelSample - rayOrigin;

        return Ray(rayOrigin, rayDirection);
    }

    __device__ Color rayColor(Ray ray, const HittableList &world, unsigned int maxDepth, hiprandState *cuRandState) {
        auto finalColor = Color(0, 0, 0);
        auto accumulatedAttenuation = Color(1, 1, 1);

        for (unsigned int depth = 0; depth < maxDepth; depth++) {
            HitRecord rec;
            if (world.hit(ray, Interval(0.001, infinity), rec)) {
                Ray scattered;
                Color attenuation;
                if (rec.material->scatter(ray, rec, attenuation, scattered, cuRandState)) {
                    ray = scattered;
                    accumulatedAttenuation = accumulatedAttenuation * attenuation;
                } else {
                    finalColor = Color(0, 0, 0);
                    break;
                }
            } else {
                const Vec3 unitDirection = unitVector(ray.direction());
                auto a = 0.5 * (unitDirection.y() + 1.0);
                finalColor = (1.0 - a) * Color(1.0, 1.0, 1.0) + a * Color(0.5, 0.7, 1.0);
                break;
            }
        }

        return accumulatedAttenuation * finalColor;
    }

    C_DH inline double linearToGamma(double linearComponent) {
        return linearComponent > 0 ? sqrt(linearComponent) : 0.0;
    }

    __global__ void writeColor(
        const HittableList &world, const Camera &camera, unsigned char *pixels, hiprandState *randStates) {
        // printf("%d", world.count);
        const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < camera.imageWidth && y < camera.imageHeight) {
            const unsigned int idx = y * camera.imageWidth + x;
            hiprandState *state = &randStates[idx];

            Color pixelColor = Color(0, 0, 0);
            for (int sample = 0; sample < camera.samplesPerPixel; sample++) {
                Ray r = getRay(x, y, camera, state);
                pixelColor += rayColor(r, world, camera.maxDepth, state);
            }
            pixelColor = pixelColor * camera.pixelSamplesScale;

            const auto r = linearToGamma(pixelColor.x());
            const auto g = linearToGamma(pixelColor.y());
            const auto b = linearToGamma(pixelColor.z());

            // Translate the [0,1] component values to the byte range [0,255].
            const Interval intensity(0.000, 0.999);
            int rByte = static_cast<int>(256 * intensity.clamp(r));
            int gByte = static_cast<int>(256 * intensity.clamp(g));
            int bByte = static_cast<int>(256 * intensity.clamp(b));

            // const char *s = "" + rByte + gByte + bByte;
            // printf("[%d;%d;%d] %f;%f;%f \n", x, y, idx, r, g, b);

            const unsigned int outputPosition = idx * 4;
            pixels[outputPosition + 0] = static_cast<unsigned char>(rByte); // R
            pixels[outputPosition + 1] = static_cast<unsigned char>(gByte); // G
            pixels[outputPosition + 2] = static_cast<unsigned char>(bByte); // B
            pixels[outputPosition + 3] = 255; // A
        }
    }

    __global__ void initCurand(hiprandState *state, unsigned long seed, int imageWidth, int imageHeight) {
        const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < imageWidth && y < imageHeight) {
            const unsigned int idx = y * imageWidth + x;
            hiprand_init(seed, idx, 0, &state[idx]);
        }
    }
}

#endif
